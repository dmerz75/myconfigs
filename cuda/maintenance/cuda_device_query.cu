#include <stdio.h>

// http://www.geforce.com/hardware/notebook-gpus/geforce-gtx-870m/specifications
// GPU Engine Specs:
// CUDA Cores                      1344
// Graphics Clock (MHz)            941 + Boost
// Memory Specs:
// Memory Clock (MHZ)              Up to 2500 MHz
// Standard Memory Configuration   GDDR5
// Memory Interface Width          192-bit
// Memory Bandwidth (GB/sec)       120.0

int main() {

    // Number of CUDA devices
    int nDevices;

    hipGetDeviceCount(&nDevices);

    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", nDevices);


    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("\n\n");
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate / 1000);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("\n\n");


        // CUDA Device Query
        /* From Nitin Gupta */
        /* http://cuda-programming.blogspot.com/2013/01/how-to-query-to-devices-in-cuda-cc.html */
        // modified
        printf("Major revision number:         %d\n",  prop.major);
        printf("Minor revision number:         %d\n",  prop.minor);
        printf("Name:                          %s\n",  prop.name);
        printf("Total global memory:           %u\n",  prop.totalGlobalMem);

        printf("Total global memory (GB):      %u\n",  prop.totalGlobalMem / 1024 / 1024);

        printf("Total shared memory per block: %u\n",  prop.sharedMemPerBlock);
        printf("Total registers per block:     %d\n",  prop.regsPerBlock);
        printf("Warp size:                     %d\n",  prop.warpSize);
        printf("Maximum memory pitch:          %u\n",  prop.memPitch);
        printf("Maximum threads per block:     %d\n",  prop.maxThreadsPerBlock);
        // for (int i = 0; i < 3; ++i)
        //     printf("Maximum dimension %d of block:  %d\n", i, prop.maxThreadsDim[i]);
        // for (int i = 0; i < 3; ++i)
        //     printf("Maximum dimension %d of grid:   %d\n", i, prop.maxGridSize[i]);
        printf("Clock rate:                    %d\n",  prop.clockRate);
        printf("Total constant memory:         %u\n",  prop.totalConstMem);
        printf("Texture alignment:             %u\n",  prop.textureAlignment);
        printf("Concurrent copy and execution: %s\n",  (prop.deviceOverlap ? "Yes" : "No"));
        printf("Number of multiprocessors:     %d\n",  prop.multiProcessorCount);
        printf("Kernel execution timeout:      %s\n",  (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));

    }
}
